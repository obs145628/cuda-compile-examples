#include "hip/hip_runtime.h"
#include "logger.hh"
#include "particle.hh"
#include <cstdio>
#include <cstdlib>

__global__
void advance_particles(float dt, Particle* arr, int len)
{
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < len)
        arr[idx].advance(dt);
}

int main(int argc, char **argv)
{
    int n = 1000000;     
    if(argc > 1)
        n = std::atoi(argv[1]);     // Number of Particles
    if(argc > 2)
        std::srand(std::atoi(argv[2])); // Random seed

    Logger logger(std::cout);
    logger << "Initialization...\n";

    Particle* arr  = new Particle[n];
    Particle* dev_arr = nullptr;
    hipMalloc(&dev_arr, n*sizeof(Particle));
    hipMemcpy(dev_arr, arr, n*sizeof(Particle), hipMemcpyHostToDevice);
    for(int i=0; i<100; i++)
    {   // Random distance each step
        float dt = (float)rand()/(float) RAND_MAX;
        advance_particles<<< 1 +  n/256, 256>>>(dt, dev_arr, n);
        hipDeviceSynchronize();
    }

    hipMemcpy(arr, dev_arr, n*sizeof(Particle), hipMemcpyDeviceToHost);
    Vector3 total_dist(0,0,0);
    Vector3 temp;
    for(int i=0; i<n; i++)
    {
        temp = arr[i].total_distance_get();
        total_dist.x += temp.x;
        total_dist.y += temp.y;
        total_dist.z += temp.z;
    }
    float avgX = total_dist.x /(float)n;
    float avgY = total_dist.y /(float)n;
    float avgZ = total_dist.z /(float)n;
    float avgNorm = std::sqrt(avgX*avgX + avgY*avgY + avgZ*avgZ);
    printf("Moved %d Particles 100 steps. Average distance traveled is |(%f, %f, %f)| = %f\n", 
                                          n, avgX, avgY, avgZ, avgNorm);
    return 0;
}
